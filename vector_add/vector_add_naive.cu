
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_kernel(float *x, float *y, float *out, int n) {
  for (int i = 0; i < n; ++i) {
    out[i] = x[i] + y[i];
  }
}

int main() {
  int N = 10000000;
  size_t mem_size = sizeof(float) * N;

  float *x, *y, *out;
  float *cuda_x, *cuda_y, *cuda_out;

  // Allocate host CPU memory for x, y
  x = static_cast<float *>(malloc(mem_size));
  y = static_cast<float *>(malloc(mem_size));

  // Initialize x = 1, y = 2
  for (int i = 0; i < N; ++i) {
    x[i] = 1.0;
    y[i] = 2.0;
  }

  // Allocate Device CUDA memory for cuda_x and cuda_y, copy them.
  hipMalloc((void **)&cuda_x, mem_size);
  hipMemcpy(cuda_x, x, mem_size, hipMemcpyHostToDevice);

  hipMalloc((void **)&cuda_y, mem_size);
  hipMemcpy(cuda_y, y, mem_size, hipMemcpyHostToDevice);

  // Allocate cuda_out CUDA memory and launch add_kernel
  hipMalloc((void **)&cuda_out, mem_size);
  add_kernel<<<1, 1>>>(cuda_x, cuda_y, cuda_out, N);

  // Copy result from GPU into CPU
  out = static_cast<float *>(malloc(mem_size));
  hipMemcpy(out, cuda_out, mem_size, hipMemcpyDeviceToHost);

  // Sync CUDA stream to wait kernel completation
  hipDeviceSynchronize();

  // Print result and checkout out = 3.
  for (int i = 0; i < 10; ++i) {
    printf("out[%d] = %.3f\n", i, out[i]);
  }

  // Free CUDA Memory
  hipFree(cuda_x);
  hipFree(cuda_y);
  hipFree(cuda_out);

  // Free Host CPU Memory
  free(x);
  free(y);
  free(out);

  return 0;
}