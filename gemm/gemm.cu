#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "configs.h"
#include "runner.cuh"
#include "utils.cuh"

std::string kernel_idx_to_name(int kernel_idx) {
  if (kernel_idx < 0 || kernel_idx >= registered_kernel.size()) {
    printf("Please enter a valid kernel number (0-%ld), valid kernels are as "
           "follows:\n",
           registered_kernel.size() - 1);
    for (int i = 0; i < registered_kernel.size(); ++i) {
      printf("Kernel %d: %s\n", i, registered_kernel[i].c_str());
    }
    exit(EXIT_FAILURE);
  }
  return registered_kernel[kernel_idx];
}

void trigger_kernel_once(const std::string &kernel_to_run, const int m,
                         const int n, const int k) {

  print_border_line();
  printf("Run kernel %s: m = %d, n = %d, k = %d\n", kernel_to_run.c_str(), m, n,
         k);

  // Allocate space for the matrices, the goal is to calculate A @ B and store
  // it in C; h for host, d for device.
  size_t size_A = m * k * sizeof(float);
  size_t size_B = k * n * sizeof(float);
  size_t size_C = m * n * sizeof(float);

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
  h_A = (float *)malloc(size_A);
  h_B = (float *)malloc(size_B);
  h_C = (float *)malloc(size_C);

  CUDA_CHECK(hipMalloc(&d_A, size_A));
  CUDA_CHECK(hipMalloc(&d_B, size_B));
  CUDA_CHECK(hipMalloc(&d_C, size_C));

  // Initialize the matrices, and copy them to device.
  randomize_matrix(h_A, m * k);
  randomize_matrix(h_B, k * n);
  zero_init_matrix(h_C, m * n);
  CUDA_CHECK(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice));

  // Run kernel and copy result from d_C back to h_C.
  bool valid_kernel = run_kernel(d_A, d_B, d_C, m, n, k, kernel_to_run);
  if (!valid_kernel)
    printf("Invalid kernel!\n");
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));

  // Free Memory.
  free(h_A);
  free(h_B);
  free(h_C);
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  print_border_line();
}

void run_tests(const std::vector<std::string> &kernels_to_run) {

  for (int test_case = 0; test_case < mnk_list.size(); ++test_case) {
    print_border_line();

    const int m = mnk_list[test_case][0];
    const int n = mnk_list[test_case][1];
    const int k = mnk_list[test_case][2];

    printf("Test %d: m = %d, n = %d, k = %d\n", test_case, m, n, k);
    estimate_compute_and_IO_cost(m, n, k, device_fp32_compute_capacity_tflops,
                                 device_global_mem_bandwidth_GB_per_sec);

    // Allocate space for the matrices, the goal is to calculate A @ B and store
    // it in C; h for host, d for device; h_C_ref is for correctness checking
    size_t size_A = m * k * sizeof(float);
    size_t size_B = k * n * sizeof(float);
    size_t size_C = m * n * sizeof(float);

    float *h_A, *h_B, *h_C, *h_C_ref, *d_A, *d_B, *d_C, *d_C_ref;
    h_A = (float *)malloc(size_A);
    h_B = (float *)malloc(size_B);
    h_C = (float *)malloc(size_C);
    h_C_ref = (float *)malloc(size_C);
    CUDA_CHECK(hipMalloc(&d_A, size_A));
    CUDA_CHECK(hipMalloc(&d_B, size_B));
    CUDA_CHECK(hipMalloc(&d_C, size_C));
    CUDA_CHECK(hipMalloc(&d_C_ref, size_C));

    // Initialize the matrices, and copy them to device.
    randomize_matrix(h_A, m * k);
    randomize_matrix(h_B, k * n);
    zero_init_matrix(h_C_ref, m * n);
    CUDA_CHECK(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_ref, h_C_ref, size_C, hipMemcpyHostToDevice));

    // Store the correct result in d_C_ref, and copy back.
    run_cublas_gemm(d_A, d_B, d_C_ref, m, n, k);
    CUDA_CHECK(hipMemcpy(h_C_ref, d_C_ref, size_C, hipMemcpyDeviceToHost));

    // Test each kernel.
    for (const std::string &kernel : kernels_to_run) {
      printf("\nKernel: %s\n", kernel.c_str());

      zero_init_matrix(h_C, m * n);
      CUDA_CHECK(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice));

      // Check kernel validity.
      bool valid_kernel = run_kernel(d_A, d_B, d_C, m, n, k, kernel);
      if (!valid_kernel)
        continue;

      // Check Correctness.
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));
      bool correct = check_result_correctness(h_C, h_C_ref, m, n);
      if (!correct)
        continue;

      // Check Performance.
      check_performance(kernel, d_A, d_B, d_C, m, n, k, 5);
    }

    // Free Memory.
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
  }
  print_border_line();
}

int main(int argc, char **argv) {

  // Get device if set in environment variable.
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }
  CUDA_CHECK(hipSetDevice(deviceIdx));

  // Mode 0: list valid kernels.
  if ((argc >= 2) && (std::string(argv[1]) == "--list-kernels")) {
    for (int i = 0; i < registered_kernel.size(); ++i) {
      printf("Kernel %d: %s\n", i, registered_kernel[i].c_str());
    }
    return 0;
  }

  // Mode 1: Trigger the kernel once without any testing.
  if ((argc >= 2) && (std::string(argv[1]) == "--once")) {
    if (argc != 6) {
      printf("Too many or too few arguments! Usage: ./gemm --once [kernel_idx] "
             "[M] [N] [K]\n");
      exit(EXIT_FAILURE);
    } else {
      CudaDeviceInfo(); // Print device information.
      std::string kernel_to_run = kernel_idx_to_name(std::stoi(argv[2]));
      int m = std::stoi(argv[3]);
      int n = std::stoi(argv[4]);
      int k = std::stoi(argv[5]);
      trigger_kernel_once(kernel_to_run, m, n, k);
    }
    return 0;
  }

  // Mode 2: Do all the tests, including correctness check and performance
  // check.
  std::vector<std::string> collected_kernels;
  if (argc == 1) {
    collected_kernels = registered_kernel;
  } else if (argc == 2) {
    int kernel_idx = std::stoi(argv[1]);
    collected_kernels.push_back(kernel_idx_to_name(kernel_idx));
  } else {
    printf("Too many arguments! Usage: ./gemm for testing of all kernels; "
           "./gemm [kernel idx] for testing one kernel.");
    exit(EXIT_FAILURE);
  }
  CudaDeviceInfo(); // Print device information.
  run_tests(collected_kernels);

  return 0;
}